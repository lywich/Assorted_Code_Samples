#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"

#include <iostream>
#include <unordered_map>

const size_t MAX_SEQ_LENGTH = 200001; 

struct DNA {
  char seq[MAX_SEQ_LENGTH];
  int seq_len;
  char qual[MAX_SEQ_LENGTH];
};

struct MatchPair {
  int sample_id;
  int signature_id;
  double match_score;
};


__global__ void matcher(DNA* d_samples, DNA* d_signatures, MatchPair* d_matches, int* d_matches_count, int N, int M) {
  __shared__ int partial_matches_count;
  __shared__ int starting_index;
  if (threadIdx.x == 0) {
    partial_matches_count = 0;
    starting_index = 0;
  }
  __syncthreads();
  
  if (blockIdx.x < N && threadIdx.x < M) {
    bool match = false;
    int thread_index = 0;
    int limit = d_samples[blockIdx.x].seq_len - d_signatures[threadIdx.x].seq_len;
    double score = 0;
    for (int j = 0; j <= limit; j++) {
      for (int k = 0; k < d_signatures[threadIdx.x].seq_len; k++) {
        if (d_samples[blockIdx.x].seq[j + k] != 'N' && d_signatures[threadIdx.x].seq[k] != 'N' && d_samples[blockIdx.x].seq[j + k] != d_signatures[threadIdx.x].seq[k]) {
          // not a match, exit loop
          score = 0;
          break;
        }
        score += d_samples[blockIdx.x].qual[j + k] - 33;
        if (k == d_signatures[threadIdx.x].seq_len - 1) {
          thread_index = atomicAdd(&partial_matches_count, 1);
          match = true;
        }
      }
      if (match) {
        break;
      }
    }
    __syncthreads();
    
    if (threadIdx.x == 0) {
      starting_index = atomicAdd(d_matches_count, partial_matches_count);
    }
    __syncthreads();
    if (match) {
      double confidence_score = score / d_signatures[threadIdx.x].seq_len;
      MatchPair pair;
      pair.sample_id = blockIdx.x;
      pair.signature_id = threadIdx.x;
      pair.match_score = confidence_score;
      d_matches[starting_index + thread_index] = pair;
    }
  }
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
  int N = samples.size();
  int M = signatures.size();

  std::vector<DNA> h_samples(N);
  std::vector<DNA> h_signatures(M);
  std::vector<MatchPair> h_matches(N * M);
  int h_matches_count = 0;

  for (size_t i = 0; i < N; i++) {
    DNA dna;
    std::string seq = samples[i].seq;
    int seq_len = seq.length();
    strncpy(dna.seq, seq.c_str(), seq_len);
    dna.seq[seq_len] = '\0'; 
    strncpy(dna.qual, samples[i].qual.c_str(), seq_len);
    dna.qual[seq_len] = '\0';
    dna.seq_len = seq_len;
    h_samples[i] = dna;
  }

  for (size_t i = 0; i < M; i++) {
    DNA dna;
    std::string seq = signatures[i].seq;
    int seq_len = seq.length();
    strncpy(dna.seq, seq.c_str(), seq_len);
    dna.seq[seq_len] = '\0'; 
    strncpy(dna.qual, signatures[i].qual.c_str(), seq_len);
    dna.qual[seq_len] = '\0'; 
    dna.seq_len = seq_len;
    h_signatures[i] = dna;
  }

  DNA* d_samples;
  DNA* d_signatures;
  MatchPair* d_matches;
  int* d_matches_count;

  hipMalloc((void**)&d_samples, N * sizeof(DNA));
  hipMalloc((void**)&d_signatures, M * sizeof(DNA));
  hipMalloc((void**)&d_matches, N * M * sizeof(MatchPair));
  hipMalloc((void**)&d_matches_count, sizeof(int));

  hipMemcpyAsync(d_samples, h_samples.data(), N * sizeof(DNA), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_signatures, h_signatures.data(),  M * sizeof(DNA), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_matches, h_matches.data(),  N * M * sizeof(MatchPair), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_matches_count, &h_matches_count,  sizeof(int), hipMemcpyHostToDevice);
  
  matcher<<<N, M>>>(d_samples, d_signatures, d_matches, d_matches_count, N, M);

  hipDeviceSynchronize();

  hipMemcpyAsync(&h_matches_count, d_matches_count,  sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync(h_matches.data(), d_matches,  h_matches_count * sizeof(MatchPair), hipMemcpyDeviceToHost);

  for (int i = 0; i < h_matches_count; i++) {
    MatchPair pair = h_matches[i];
    MatchResult match_result;
    match_result.sample_name = samples[pair.sample_id].name;
    match_result.signature_name = signatures[pair.signature_id].name;
    match_result.match_score = pair.match_score;
    matches.push_back(match_result);
  }

  hipFree(d_samples);
  hipFree(d_signatures);
  hipFree(d_matches);
  hipFree(d_matches_count);
}
